#include "hip/hip_runtime.h"
#include "hip/hip_runtime_api.h"
#include "stdio.h"
#include <iostream>

#include "utils.hpp"

/* matmul的函数实现*/
__global__ void MatmulKernel(float *M_device, float *N_device, float *P_device, int width){
    /* 
        我们设定每一个thread负责P中的一个坐标的matmul
        所以一共有width * width个thread并行处理P的计算
    */
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    float P_element = 0;

    /* 对于每一个P的元素，我们只需要循环遍历width次M和N中的元素就可以了*/
    for (int k = 0; k < width; k ++){
        float M_element = M_device[y * width + k];
        float N_element = N_device[k * width + x];
        P_element += M_element * N_element;
    }

    P_device[y * width + x] = P_element;
}

/*

    这个实现的问题点：只有一个block
    因为只有一个block，并且又因为SM中的sp数量是有限的，所以不能够全部放下。想要全部放下的话需要缩小矩阵的大小
    有很多次读写，但具体的执行很少(两次读和一次写，一次计算)
    解决办法：使用tile
*/
void MatmulOnDevice(float *M_host, float *N_host, float* P_host, int width, int blockSize){
    /* 设置矩阵大小 */
    int size = width * width * sizeof(float);

    /* 分配M, N在GPU上的空间*/
    float *M_device;
    float *N_device;
    CUDA_CHECK(hipMalloc(&M_device, size));
    CUDA_CHECK(hipMalloc(&N_device, size));

    /* 分配M, N拷贝到GPU上*/
    CUDA_CHECK(hipMemcpy(M_device, M_host, size, hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(N_device, N_host, size, hipMemcpyHostToDevice));

    /* 分配P在GPU上的空间*/
    float *P_device;
    CUDA_CHECK(hipMalloc(&P_device, size));

    /* 调用kernel来进行matmul计算, 在这个例子中我们用的方案是：使用一个grid，一个grid里有width*width个线程 */
    dim3 dimBlock(blockSize, blockSize);
    dim3 dimGrid(width / blockSize, width / blockSize);
    MatmulKernel <<<dimGrid, dimBlock>>> (M_device, N_device, P_device, width);

    /* 将结果从device拷贝回host*/
    CUDA_CHECK(hipMemcpy(P_host, P_device, size, hipMemcpyDeviceToHost));
    CUDA_CHECK(hipDeviceSynchronize());

    /* 注意要在synchronization结束之后排查kernel的错误 */
    LAST_KERNEL_CHECK(); 

    /* Free */
    hipFree(P_device);
    hipFree(N_device);
    hipFree(M_device);
}

