#include "hip/hip_runtime.h"

#include "stdio.h"

/* matmul的函数实现*/
__global__ void MatmulKernel(float *M_device, float *N_device, float *P_device, int width){
    /* 
        我们设定每一个thread负责P中的一个坐标的matmul
        所以一共有width * width个thread并行处理P的计算
    */
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    float P_element = 0;

    /* 对于每一个P的元素，我们只需要循环遍历width次M和N中的元素就可以了*/
    for (int k = 0; k < width; k ++){
        float M_element = M_device[y * width + k];
        float N_element = N_device[k * width + x];
        P_element += M_element * N_element;
    }

    P_device[y * width + x] = P_element;
}

/*
    CUDA中使用block对矩阵中某一片区域进行集中计算。这个类似于loop中的tile
    感兴趣的同学可以试着改一下blockSize，也就是tileSize，看看速度会发生什么样子的变化
    当blockSize达到一个数量的时候，这个程序会出错。下一个案例中我们会分析
*/
void MatmulOnDevice(float *M_host, float *N_host, float* P_host, int width, int blockSize){
    /* 设置矩阵大小 */
    int size = width * width * sizeof(float);

    /* 分配M, N在GPU上的空间*/
    float *M_device;
    float *N_device;

    hipMalloc(&M_device, size);
    hipMalloc(&N_device, size);

    /* 分配M, N拷贝到GPU上*/
    hipMemcpy(M_device, M_host, size, hipMemcpyHostToDevice);
    hipMemcpy(N_device, N_host, size, hipMemcpyHostToDevice);

    /* 分配P在GPU上的空间*/
    float *P_device;
    hipMalloc(&P_device, size);

    /* 调用kernel来进行matmul计算, 在这个例子中我们用的方案是：将一个矩阵切分成多个blockSize * blockSize的大小 */
    dim3 dimBlock(blockSize, blockSize);
    dim3 dimGrid(width / blockSize, width / blockSize);
    MatmulKernel <<<dimGrid, dimBlock>>> (M_device, N_device, P_device, width);

    /* 将结果从device拷贝回host*/
    hipMemcpy(P_host, P_device, size, hipMemcpyDeviceToHost);
    hipDeviceSynchronize();

    /* Free */
    hipFree(P_device);
    hipFree(N_device);
    hipFree(M_device);
}

