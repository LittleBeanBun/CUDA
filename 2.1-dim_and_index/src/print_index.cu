#include <hip/hip_runtime.h>
#include <stdio.h>


__global__ void print_idx_kernel(){
    printf("block idx: (%3d, %3d, %3d), thread idx: (%3d, %3d, %3d)\n",
         blockIdx.z, blockIdx.y, blockIdx.x,
         threadIdx.z, threadIdx.y, threadIdx.x);
}

__global__ void print_dim_kernel(){
    printf("grid dimension: (%3d, %3d, %3d), block dimension: (%3d, %3d, %3d)\n",
         gridDim.z, gridDim.y, gridDim.x,
         blockDim.z, blockDim.y, blockDim.x);
}

__global__ void print_thread_idx_per_block_kernel(){
    int index = threadIdx.z * blockDim.x * blockDim.y + \
              threadIdx.y * blockDim.x + \
              threadIdx.x;

    printf("block idx: (%3d, %3d, %3d), thread idx: %3d\n",
         blockIdx.z, blockIdx.y, blockIdx.x,
         index);
}

__global__ void print_thread_idx_per_grid_kernel(){
    int bSize  = blockDim.z * blockDim.y * blockDim.x;

    int bIndex = blockIdx.z * gridDim.x * gridDim.y + \
               blockIdx.y * gridDim.x + \
               blockIdx.x;

    int tIndex = threadIdx.z * blockDim.x * blockDim.y + \
               threadIdx.y * blockDim.x + \
               threadIdx.x;

    int index  = bIndex * bSize + tIndex;

    printf("block idx: %3d, thread idx in block: %3d, thread idx: %3d\n", 
         bIndex, tIndex, index);
}

__global__ void print_cord_kernel(){
    int index = threadIdx.z * blockDim.x * blockDim.y + \
              threadIdx.y * blockDim.x + \
              threadIdx.x;

    int x  = blockIdx.x * blockDim.x + threadIdx.x;
    int y  = blockIdx.y * blockDim.y + threadIdx.y;

    printf("block idx: (%3d, %3d, %3d), thread idx: %3d, cord: (%3d, %3d)\n",
         blockIdx.z, blockIdx.y, blockIdx.x,
         index, x, y);
}

void print_one_dim(){
    int inputSize = 8;
    int blockDim = 4;
    int gridDim = inputSize / blockDim;

    dim3 block(blockDim);
    dim3 grid(gridDim);

    /* 这里建议大家吧每一函数都试一遍*/
    // print_idx_kernel<<<grid, block>>>();
    // print_dim_kernel<<<grid, block>>>();
    // print_thread_idx_per_block_kernel<<<grid, block>>>();
    print_thread_idx_per_grid_kernel<<<grid, block>>>();

    hipDeviceSynchronize();
}

void print_two_dim(){
    int inputWidth = 4;

    int blockDim = 2;
    int gridDim = inputWidth / blockDim;

    dim3 block(blockDim, blockDim);
    dim3 grid(gridDim, gridDim);

    /* 这里建议大家吧每一函数都试一遍*/
    // print_idx_kernel<<<grid, block>>>();
    // print_dim_kernel<<<grid, block>>>();
    // print_thread_idx_per_block_kernel<<<grid, block>>>();
    print_thread_idx_per_grid_kernel<<<grid, block>>>();

    hipDeviceSynchronize();
}

void print_cord(){
    int inputWidth = 4;

    int blockDim = 2;
    int gridDim = inputWidth / blockDim;

    dim3 block(blockDim, blockDim);
    dim3 grid(gridDim, gridDim);

    print_cord_kernel<<<grid, block>>>();

    hipDeviceSynchronize();
}

int main() {
    /*
    synchronize是同步的意思，有几种synchronize

    cudaDeviceSynchronize: CPU与GPU端完成同步，CPU不执行之后的语句，知道这个语句以前的所有cuda操作结束
    cudaStreamSynchronize: 跟cudaDeviceSynchronize很像，但是这个是针对某一个stream的。只同步指定的stream中的cpu/gpu操作，其他的不管
    cudaThreadSynchronize: 现在已经不被推荐使用的方法
    __syncthreads:         线程块内同步
    */
    // print_one_dim();
    // print_two_dim();
    print_cord();
    return 0;
}
