#include "hip/hip_runtime.h"
#include "hip/hip_runtime_api.h"
#include "stdio.h"
#include <iostream>

#include "utils.hpp"

template <typename T>
__global__ void resize_bilinear_BGR2RGB_shift_kernel(
    T* tar, uint8_t* src, 
    int tarW, int tarH, 
    int srcW, int srcH, 
    float scaled_w, float scaled_h) 
{

    // resized之后的图tar上的坐标
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    // bilinear interpolation -- 计算x,y映射到原图时最近的4个坐标
    int src_y1 = round((float)y * scaled_h);
    int src_x1 = round((float)x * scaled_w);
    int src_y2 = src_y1 + 1;
    int src_x2 = src_x1 + 1;

    if (src_y1 < 0 || src_x1 < 0 || src_y1 > srcH || src_x1 > srcW) {
        // bilinear interpolation -- 对于越界的坐标不进行计算
    } else {
        // bilinear interpolation -- 计算原图上的坐标(浮点类型)在0~1之间的值
        float th   = (float)y * scaled_h - src_y1;
        float tw   = (float)x * scaled_w - src_x1;

        // bilinear interpolation -- 计算面积(这里建议自己手画一张图来理解一下)
        float a1_1 = (1.0 - tw) * (1.0 - th);
        float a1_2 = (1.0 - tw) * th;
        float a2_1 = tw * (1.0 - th);
        float a2_2 = tw * th;

        // bilinear interpolation -- 计算4个坐标所对应的索引
        int srcIdx1_1 = (src_y1 * srcW + src_x1) * 3;
        int srcIdx1_2 = (src_y1 * srcW + src_x2) * 3;
        int srcIdx2_1 = (src_y2 * srcW + src_x1) * 3;
        int srcIdx2_2 = (src_y2 * srcW + src_x2) * 3;

        // bilinear interpolation -- 计算原图在目标图中的x, y方向上的偏移量
        y = y - int(srcH / (scaled_h * 2)) + int(tarH / 2);
        x = x - int(srcW / (scaled_w * 2)) + int(tarW / 2);

        // bilinear interpolation -- 计算resized之后的图的索引
        int tarIdx    = (y * tarW  + x) * 3;

        // bilinear interpolation -- 实现bilinear interpolation + BGR2RGB
        tar[tarIdx + 0] = round(
                          a1_1 * src[srcIdx1_1 + 2] + 
                          a1_2 * src[srcIdx1_2 + 2] +
                          a2_1 * src[srcIdx2_1 + 2] +
                          a2_2 * src[srcIdx2_2 + 2]);

        tar[tarIdx + 1] = round(
                          a1_1 * src[srcIdx1_1 + 1] + 
                          a1_2 * src[srcIdx1_2 + 1] +
                          a2_1 * src[srcIdx2_1 + 1] +
                          a2_2 * src[srcIdx2_2 + 1]);

        tar[tarIdx + 2] = round(
                          a1_1 * src[srcIdx1_1 + 0] + 
                          a1_2 * src[srcIdx1_2 + 0] +
                          a2_1 * src[srcIdx2_1 + 0] +
                          a2_2 * src[srcIdx2_2 + 0]);
    }
}

template <typename T>
void resize_bilinear_gpu(
    T* d_tar, uint8_t* d_src, 
    int tarW, int tarH, 
    int srcW, int srcH)
{
    dim3 dimBlock(16, 16, 1);
    dim3 dimGrid(tarW / 16 + 1, tarH / 16 + 1, 1);
    
    //scaled resize
    float scaled_h = (float)srcH / tarH;
    float scaled_w = (float)srcW / tarW;
    float scale = (scaled_h > scaled_w ? scaled_h : scaled_w);

    scaled_h = scale;
    scaled_w = scale;
    
    resize_bilinear_BGR2RGB_shift_kernel <<<dimGrid, dimBlock>>> (d_tar, d_src, tarW, tarH, srcW, srcH, scaled_w, scaled_h);
}

template __global__ void resize_bilinear_BGR2RGB_shift_kernel<uint8_t>(uint8_t* tar, uint8_t* src, int tarW, int tarH, int srcW, int srcH, float scaled_w, float scaled_h);
template __global__ void resize_bilinear_BGR2RGB_shift_kernel<float>(float* tar, uint8_t* src, int tarW, int tarH, int srcW, int srcH, float scaled_w, float scaled_h);
template void resize_bilinear_gpu<uint8_t>(uint8_t* d_tar, uint8_t* d_src, int tarW, int tarH, int srcW, int srcH);
template void resize_bilinear_gpu<float>(float* d_tar, uint8_t* d_src, int tarW, int tarH, int srcW, int srcH);
